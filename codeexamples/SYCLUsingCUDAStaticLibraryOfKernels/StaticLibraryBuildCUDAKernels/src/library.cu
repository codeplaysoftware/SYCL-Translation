#include "hip/hip_runtime.h"
// In-house headers:
#include "staticlibfnscudakernelsDXTC.h"
#include "cudakernels.cu"

namespace StaticLibFnsCUDAKernelsDXTC
{
    hipError_t CompressFacade( const int vBlocksPerLaunch, const uint vBlocks, 
                                const uint *vpPermutations, const uint *vpImage, uint *vpResult, const int vBlockOffset ) 
    {
        compress<<< min( vBlocksPerLaunch, vBlocks - vBlockOffset ), NUM_THREADS >>>(
              vpPermutations, vpImage, (uint2 *) vpResult, vBlockOffset );

        // Interop with host_task doesn't add CUDA event to task graph
        // so we must manually sync here.
        hipDeviceSynchronize();

        return hipGetLastError();
    }

} // namespace StaticLibFnsCUDAKernelsDXTC
